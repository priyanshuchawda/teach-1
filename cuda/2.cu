
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function to add two numbers
__global__ void add(int a, int b, int *result) {
    *result = a + b;
}

int main() {
    int a = 5, b = 7;
    int *d_result, h_result;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_result, sizeof(int));

    // Launch kernel
    add<<<1, 1>>>(a, b, d_result);

    // Copy result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: " << h_result << std::endl;

    // Free GPU memory
    hipFree(d_result);

    return 0;
}
